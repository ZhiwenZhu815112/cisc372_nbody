#include "hip/hip_runtime.h"
/********************************************/
/*Project authors: Jingqing Liu, Zhiwen Zhu*/
/******************************************/
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

	 vector3 *device_hPos, *device_hVel, *device_accels, *device_accel_sum;
	 double *device_mass;

//first compute the pairwi  se accelerations.  Effect is on the first argument.
__global__ void compute_Pairwise_Accelerations(vector3 *hPos, double *mass, vector3 *accels) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NUMENTITIES && j < NUMENTITIES) {
		if (i == j) {
			FILL_VECTOR(accels[i * NUMENTITIES + j], 0, 0, 0);
		} else {
			vector3 distance;
			for (int k = 0; k < 3; k++) {
				distance[k] = hPos[i][k] - hPos[j][k];
			}
			double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
			double magnitude = sqrt(magnitude_sq);
			double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
			FILL_VECTOR(accels[i * NUMENTITIES + j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
		}
	} 
}

// __global__ void sum(vector3 *accels, vector3 *accel_sum, int numEntities) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < numEntities) {
//         FILL_VECTOR(accel_sum[i], 0, 0, 0);
//         for (int j = 0; j < numEntities; j++) {
//             for (int k = 0; k < 3; k++) {
//                 accel_sum[i][k] += accels[i * numEntities + j][k];
//             }
//         }
//     }
// }
// __global__ void update_velocity_and_position(vector3 *hPos, vector3 *hVel, vector3 * accel_sum, int numEntities) {

// 	int i = blockIdx.x * blockDim.x + threadIdx.x;

// 	if (i < numEntities) {
// 		for (int k = 0; k < 3; k++){
// 			hVel[i][k] += accel_sum[i][k] * INTERVAL;
// 			hPos[i][k] = hVel[i][k] * INTERVAL;
// 		}
// 	}
// }

// instead of spilt sum and update into two functions, the combination of two functions actually speed up little bit. 
 
__global__ void sum_and_update_velocity_and_position(vector3* hPos, vector3* hVel, vector3* accels, int numEntities) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < numEntities) {
		vector3 accel_sum={0, 0, 0};
		for (int j = 0; j < numEntities; j++){
			for (int k = 0;k < 3; k++) {
				accel_sum[k] += accels[i * numEntities + j][k];
			}
		}

	//compute the new velocity based on the acceleration and time interval
	//compute the new position based on the velocity and time interval
		for (int k = 0; k < 3; k++){
			hVel[i][k] += accel_sum[k] * INTERVAL;
			hPos[i][k] = hVel[i][k] * INTERVAL;
		}
	}
}



//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){


	dim3 blockDim(16, 16);
	dim3 gridDim((NUMENTITIES + blockDim.x - 1) / blockDim.x, (NUMENTITIES + blockDim.y - 1) / blockDim.y);

	compute_Pairwise_Accelerations<<<gridDim, blockDim>>>(device_hPos, device_mass, device_accels);
	hipDeviceSynchronize();

	// sum<<<gridDim.x, blockDim.x>>>(device_accels, device_accel_sum, NUMENTITIES);
    // hipDeviceSynchronize();
	
	// update_velocity_and_position<<<gridDim.x, blockDim.x>>>(device_hPos, device_hVel, device_accel_sum, NUMENTITIES);

	sum_and_update_velocity_and_position<<<gridDim.x, blockDim.x>>>(device_hPos, device_hVel, device_accels,NUMENTITIES);



	hipMemcpy(hPos, device_hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hVel, device_hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);

}